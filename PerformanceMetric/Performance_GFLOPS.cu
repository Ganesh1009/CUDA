
#include <hip/hip_runtime.h>
#include <stdio.h>

#define n 1024*1024


__global__ void kernel(int a,int *x, int *y)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
		y[i]=a*x[i]+y[i];

}

int main(void)
{
  float elapsedTime = 0.0f;
  hipEvent_t start, stop;
  hipError_t err=hipSuccess;
  int *host_a,*host_b;
  host_b = (int *) malloc(n);
  host_a = (int *) malloc(n);
  int *dev_array_a,*dev_array_b;

  hipMalloc((void **)&dev_array_a, n);
  hipMalloc((void **)&dev_array_b, n);

  for (int i = 0; i < n/sizeof(int); i++) {
    host_a[i] = i;
  }
  hipMemcpy(dev_array_a, host_a, n, hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  kernel<<<256,1024>>>(2,dev_array_a,dev_array_b);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop); //Wait till the event is executed.
  hipMemcpy(host_b, dev_array_b, n, hipMemcpyDeviceToHost);
  hipEventElapsedTime(&elapsedTime,start,stop);
  printf("Time for kernel to exexute:%fms\n",elapsedTime);
  printf("Arithmetic Performance = %5f Gflops/s\n\n", n * 1e-6/elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


}
