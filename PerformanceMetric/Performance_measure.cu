#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define ONE_MB 1024*1024
#define SIXTEEN_MB 16*1024*1024
#define FOUR_MB 4*1024*1024



__global__ void testKernel1 (){

}

__global__ void testKernel2 (int *array1, int *resultArray){

  int index = threadIdx.x + blockIdx.x*blockDim.x;
  resultArray[index] = array1[index];
}

__global__ void testKernel3(int *c_array, int *transpose_array, int width){

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  transpose_array[((int)(index/width) + ((index%width)*(width)))] = c_array[index];
}


int main() {
  float time_in_ms = 0.0f;
  hipEvent_t start, stop;
  hipError_t err=hipSuccess;
  //===========TASK 1============================
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  testKernel1<<<1,1>>>();
  hipEventRecord(stop,0);
  hipEventSynchronize(stop); //Wait till the event is executed.
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Time for empty kernel to exexute:%fms\n\n",time_in_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  //========TASK 1 end============================

  //=======TASK 2=================================
  int *array1 = (int *) malloc(ONE_MB);
  int *c_array;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int i = 0; i < ONE_MB/sizeof(int); i++) {
    array1[i] = i;
  }

  hipMalloc((void **)&c_array, ONE_MB);
  hipEventRecord(start,0);
  hipMemcpy(c_array, array1, ONE_MB, hipMemcpyHostToDevice);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Time for memory copy of %d bytes is :%fms\n\n",ONE_MB,time_in_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(c_array);
  //==============TASK 2 end======================

  //=============TASK 3===========================
  int *arrayCopy = (int *) malloc(ONE_MB),*c_arrayCopy;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc((void **)&c_array, ONE_MB);
  hipMalloc((void **)&c_arrayCopy, ONE_MB);
  hipMemcpy(c_array, array1, ONE_MB, hipMemcpyHostToDevice);
  //testKernel2<<<256,1024>>>(c_array,c_arrayCopy);
  hipEventRecord(start,0);
  hipMemcpy(arrayCopy, c_arrayCopy, ONE_MB, hipMemcpyDeviceToHost);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  time_in_ms = 0.0f;
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Time to copy %d bytes from device to host = %fms\n",ONE_MB,time_in_ms);
  printf("Effective bandwidth = %f GB/s \n\n", ONE_MB/time_in_ms/1e6);
  hipFree(c_array);
  hipFree(c_arrayCopy);
  free(array1);
  free(arrayCopy);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  //============TASK 3 end==================================

  //============TASK 4====================================
  arrayCopy = (int *) malloc(SIXTEEN_MB);
  array1 = (int *) malloc(SIXTEEN_MB);
  for (int i = 0; i < (SIXTEEN_MB)/sizeof(int); i++) {
    array1[i] = i;
  }
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc((void **)&c_array, 16*ONE_MB);
  hipMalloc((void **)&c_arrayCopy, 16*ONE_MB);
  hipEventRecord(start,0);
  hipMemcpy(c_array, array1, SIXTEEN_MB, hipMemcpyHostToDevice);
  //testKernel2<<<4096,1024>>>(c_array,c_arrayCopy);
  hipMemcpy(arrayCopy, c_arrayCopy, SIXTEEN_MB, hipMemcpyDeviceToHost);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  time_in_ms = 0.0f;
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Time to copy %d bytes from device to host = %fms\n",SIXTEEN_MB,time_in_ms);
  printf("Effective bandwidth = %f GB/s \n\n", (SIXTEEN_MB)/time_in_ms/1e6);
  hipFree(c_array);
  hipFree(c_arrayCopy);
  free(array1);
  free(arrayCopy);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  //===================TASK 4 end========================

  //===================TASK 5============================
  int *c_array1,*c_array2, number_of_steams = 2;
  int *resultArray1, *resultArray2;

  hipHostMalloc((void **)&resultArray1, SIXTEEN_MB);
  hipHostMalloc((void **)&resultArray2, SIXTEEN_MB);
  hipHostMalloc((void **)&array1, SIXTEEN_MB);

  hipMalloc((void **)&c_array1, SIXTEEN_MB);
  hipMalloc((void **)&c_array2, SIXTEEN_MB);

  for (int i = 0; i < (SIXTEEN_MB)/sizeof(int); i++) {
    array1[i] = i;
  }
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream[number_of_steams];
  for (int i = 0; i < number_of_steams; i++) {
    hipStreamCreate(&stream[i]);
  }
  hipEventRecord(start,0);
  //for(int i=0;i<number_of_steams;i++){
    hipMemcpyAsync(c_array1,array1,SIXTEEN_MB,hipMemcpyHostToDevice,stream[0]);
    hipMemcpyAsync(c_array2,array1,SIXTEEN_MB,hipMemcpyHostToDevice,stream[1]);
  //}
  hipMemcpyAsync(resultArray1,c_array1,SIXTEEN_MB,hipMemcpyDeviceToHost,stream[0]);
  hipMemcpyAsync(resultArray2,c_array2,SIXTEEN_MB,hipMemcpyDeviceToHost,stream[1]);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  time_in_ms = 0.0;
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Time for async data transfer of 2 arrays each of %d bytes multiple arrays is = %fms\n",SIXTEEN_MB, time_in_ms);
  printf("Bandwidth for async data transfer of 2 arrays is : %f \n\n",(SIXTEEN_MB)/time_in_ms/1e6);
  hipFree(c_array1);
  hipFree(c_array2);
  hipHostFree(resultArray1);
  hipHostFree(resultArray2);
  hipHostFree(array1);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //==================TASK 5 end=========================


  //=================TASK 6================================
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int *array4mb = (int *) malloc(FOUR_MB*sizeof(int));
  int width = 1024;
  for(int i=0; i<FOUR_MB; i++){
    array4mb[i] = i;
  }

  int *cuda_array_4mb, *transpose_4MB;
  hipMalloc((void **)&cuda_array_4mb,FOUR_MB*(sizeof(int)));
  hipMalloc((void **)&transpose_4MB,FOUR_MB*(sizeof(int)));
  hipMemcpy(cuda_array_4mb, array4mb, FOUR_MB*(sizeof(int)), hipMemcpyHostToDevice);
  hipEventRecord(start,0);
  testKernel3<<<4096,1024>>>(cuda_array_4mb, transpose_4MB,width);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipMemcpy(array4mb, transpose_4MB, FOUR_MB*(sizeof(int)), hipMemcpyDeviceToHost);
  time_in_ms = 0.0;
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Bandwidth for array of dimension %d is %fGFLOPS\n\n",FOUR_MB,(FOUR_MB)/time_in_ms/1e6);
  //=================Task 6 end============================



  err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

  hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties,0);
  printf("Device name: %s\n", deviceProperties.name);
  printf("Memory Clock Rate (KHz): %d\n",deviceProperties.memoryClockRate);
  printf("Memory Bus Width (bits): %d\n",deviceProperties.memoryBusWidth);
  printf("Peak Memory Bandwidth (GB/s): %f\n",
    2.0*deviceProperties.memoryClockRate*(deviceProperties.memoryBusWidth/8)/1.0e6);
  printf("Registers per multiprocessors: %d\n\n",deviceProperties.regsPerMultiprocessor);
}
